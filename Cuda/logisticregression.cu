#include "hip/hip_runtime.h"
#include <eigen3/Eigen/Core>
#include "logisticregression.cuh"


// Should take in datapoints as eigen matrix (N x 3) input
Eigen::MatrixXf runLogisticRegression(const Eigen::MatrixXd &points,
                                      const Eigen::VectorXi &occupancy,
									  float learningRate,
									  float regularisationLambda)
{
	Eigen::MatrixXf tmp(1,1);
	return tmp;	
}

int getNumBlocks(int numDataPoints)
{
	return (int) ceil((float) numDataPoints / (double) MAX_THREADS);
}

void convertEigenInputToPointers(const Eigen::MatrixXf &points,
                                 const Eigen::MatrixXi &occupancy,
                                 float *h_pointsX,
                                 float *h_pointsY,
                                 float *h_pointsZ,
                                 int   *h_occupancy)
{
	// TODO: Add some kind of size checking here
	// Assumes occupancy is a row vector
	// Assumes points is N * 3 matrix (N rows, 3 cols - 1 for each coord)
	float *fullPointsOutput = (float *) malloc(sizeof(float) * points.rows() * points.cols());
	int *occupancyOutput = (int *) malloc(sizeof(int) * occupancy.cols());

	// Allocate space for the output
	h_pointsX = (float *) malloc(points.rows() * sizeof(float));
	h_pointsY = (float *) malloc(points.rows() * sizeof(float));
	h_pointsZ = (float *) malloc(points.rows() * sizeof(float));

	// Coordinate copy
	std::copy(fullPointsOutput + 0 * points.rows(), 
			  fullPointsOutput + 1 * points.rows(), 
			  h_pointsX);
	std::copy(fullPointsOutput + 1 * points.rows(), 
			  fullPointsOutput + 2 * points.rows(), 
			  h_pointsY);
	std::copy(fullPointsOutput + 2 * points.rows(), 
			  fullPointsOutput + 3 * points.rows(), 
			  h_pointsZ);

	// Occupancy copy
	std::copy(occupancyOutput, 
			  occupancyOutput + occupancy.rows(), 
			  h_occupancy);	
	
}

Eigen::MatrixXf convertWeightPointerToEigen(float *h_weights, int nWeights)
{
	return Eigen::Map<Eigen::MatrixXf>(h_weights, 1, nWeights);
}


__global__ void cudaRbf(float *d_x, float *d_y, float *d_z,
                        float *outputFeatures, int *d_pointIdx,
                        float *d_lengthScale)
{
    int cudaIdx = threadIdx.x + blockIdx.x * blockDim.x;

    float diff = (d_x[cudaIdx] - d_x[*d_pointIdx]) * (d_x[cudaIdx] - d_x[*d_pointIdx]) +
                 (d_y[cudaIdx] - d_y[*d_pointIdx]) * (d_y[cudaIdx] - d_y[*d_pointIdx]) +
                 (d_z[cudaIdx] - d_z[*d_pointIdx]) * (d_z[cudaIdx] - d_z[*d_pointIdx]);

    //outputFeatures[cudaIdx] = (float) exp(-*d_lengthScale * diff);
    outputFeatures[cudaIdx] = diff;
}

__global__ void cudaSgd(int *d_occupancy,
                        float *d_weights,
                        float *d_features,
                        int *d_pointIdx,
                        hiprandState_t *states)
{
    int cudaIdx = threadIdx.x + blockIdx.x * blockDim.x;

    // If this is is the first example, just initialise the weights
//  if (*d_pointIdx == 0) {
        // Random value between 0 and 1
//      d_weights[cudaIdx] = (float) (hiprand(&states[blockIdx.x]) % 1000) / 1000.0; 
//  } else {
        float learningRate = 0.0001;
        float lambda = 5;

        float numerator = -d_occupancy[*d_pointIdx] * d_features[cudaIdx];
    //  float denominator = 1 + exp(-numerator*d_weights[cudaIdx]);         
        float denominator = 1;

        // Just using L2 regularisation here, may use elastic net later
        float regulariser = lambda*d_weights[cudaIdx];

        // Combine all the parts
        float lossGradient = (numerator/denominator) + regulariser;

        // Update weight
        d_weights[cudaIdx] = d_weights[cudaIdx] - learningRate*lossGradient;
//  }
}

